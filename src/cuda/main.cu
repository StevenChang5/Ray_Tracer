#include "hip/hip_runtime.h"
#include <iostream>

#include "utils.hu"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line){
    if(result){
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(vec3 *fb, int max_x, int max_y){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    fb[pixel_index] = vec3(float(i)/max_x, float(j)/max_y, 0.2f);
}

int main(){
    int nx = 1200;
    int ny = 800;
    int tx = 8;
    int ty = 8;

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);

    // Allocate Frame Buffer
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for(int j = ny-1; j >= 0; j--){
        for(int i = 0; i < nx; i++){
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].x());
            int ig = int(255.99*fb[pixel_index].y());
            int ib = int(255.99*fb[pixel_index].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    checkCudaErrors(hipFree(fb));
}